#include "hip/hip_runtime.h"
#include "t20049.cuh"

namespace clanguml {
namespace t20049 {

constexpr unsigned long N{1000};

__device__ float square(float a) { return a * a; }

__global__ void vector_square_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++) {
        out[i] = add(square(a[i]), square(b[i]));
    }
}

int tmain()
{
    float *a, *b, *out;

    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    vector_square_add<<<1, 1>>>(out, a, b, N);

    return 0;
}

}
}